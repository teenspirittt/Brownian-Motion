#include "hip/hip_runtime.h"
#include <glut.h>
#include <math.h>
#include <stdio.h>
#include <windows.h>

#include <iostream>

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include ""

#define MAX_FLAKES 3000
#define SPEED 0.09
#define MAX_X 1280
#define MAX_Y 720
#define MIN_X -640
#define MIN_Y -360

void drawPoint();
void spawnPoint();
void movePoint(int i);

typedef struct snowFlake {
  float posX;
  float posY;
  float destX;
  float destY;
} snowFlake;

snowFlake* flake = NULL;
snowFlake* dev_flake = NULL;

__global__ void kernel(unsigned int seed, snowFlake* flake) {
  unsigned i = threadIdx.x + blockIdx.x * blockDim.x;

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hiprandState_t state;
  hiprand_init(seed + i, 0, 0, &state);

  int randX = MIN_X + hiprand(&state) % MAX_X;
  int randY = MIN_Y + hiprand(&state) % MAX_Y;

  if (flake[i].posX >= flake[i].destX - 20 &&
      flake[i].posX <= flake[i].destX + 20 &&
      flake[i].posY >= flake[i].destY - 20 &&
      flake[i].posY <= flake[i].destY + 20) {
    // flake[i].destX = flake[(i + 1) % MAX_FLAKES].posY;
    // flake[i].destY = flake[(i + 1) % MAX_FLAKES].posY;
    flake[i].destX = randX;
    flake[i].destY = randY;
  } else {
    double dx = (flake[i].destX - flake[i].posX) / 10.0;
    double dy = (flake[i].destY - flake[i].posY) / 10.0;
    flake[i].posX = flake[i].posX + dx * SPEED;
    flake[i].posY = flake[i].posY + dy * SPEED;
  }
}

void drawPoint() {
  glPushMatrix();
  glEnable(GL_POINT_SMOOTH);
  glPointSize(1);
  glClear(GL_COLOR_BUFFER_BIT);
  glBegin(GL_POINTS);
  glColor3f(1, 1, 1);

  hipMemcpy(dev_flake, flake, sizeof(snowFlake) * MAX_FLAKES,
             hipMemcpyHostToDevice);
  kernel<<<5, 1000>>>(time(NULL), dev_flake);
  hipMemcpy(flake, dev_flake, sizeof(snowFlake) * MAX_FLAKES,
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < MAX_FLAKES; ++i) {
    glColor3f(0, (float)(flake[i].posX + 640) / 1280,
              1 - (float)(flake[i].posX + 640) / 1280);
    glVertex2f(flake[i].posX, (flake[i].posY));
  }

  glEnd();
  glPopMatrix();
  glutSwapBuffers();
}

void spawnPoint() {
  glClearColor(0, 0, 0, 0);
  glScalef((float)1 / 640, (float)1 / 360, 1);
  glMatrixMode(GL_PROJECTION);
  flake = (snowFlake*)malloc(sizeof(snowFlake) * MAX_FLAKES);
  hipMalloc((void**)&dev_flake, sizeof(snowFlake) * MAX_FLAKES);

  for (int i = 0; i < MAX_FLAKES; ++i) {
    flake[i].posX = (-640 + rand() % 1280);
    flake[i].posY = (-360 + rand() % 720);
    // flake[i].destX = (-640 + rand() % 1280);
    // flake[i].destY = (-640 + rand() % 1280);
    flake[i].destX = 0;
    flake[i].destY = 0;
  }
  hipMemcpy(dev_flake, flake, sizeof(snowFlake) * MAX_FLAKES,
             hipMemcpyHostToDevice);
  glLoadIdentity();
  glMatrixMode(GL_MODELVIEW);
}

void timer(int value) {
  glutPostRedisplay();
  glutTimerFunc(30, timer, 0);
}

int main(int argc, char** argv) {
  glutInit(&argc, argv);  // Setting up OpenGL
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
  glutInitWindowSize(1280, 720);
  glutInitWindowPosition(400, 150);
  glutCreateWindow("takov put'");
  glutDisplayFunc(drawPoint);
  glutTimerFunc(30, timer, 0);
  spawnPoint();
  glutMainLoop();
  free(flake);
  hipFree(dev_flake);
}
